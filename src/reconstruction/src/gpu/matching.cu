#include "hip/hip_runtime.h"
#include "cudaSift.h"
#include "cudautils.h"

//================= Device matching functions =====================//

__global__ void MatchSiftPoints(SiftPoint *sift1, SiftPoint *sift2, float *corrData, int numPts1, int numPts2)
{
  __shared__ float siftPoint[128];
  __shared__ float sums[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int p1 = blockIdx.x;
  const int p2 = blockIdx.y*16 + ty;
  const float *ptr1 = sift1[p1].data;
  const float *ptr2 = sift2[p2].data;
  const int i = ty*16 + tx;
  if (ty<8)
    siftPoint[i] = ptr1[i];
  __syncthreads();
  float sum = 0.0f;
  if (p2<numPts2)
    for (int j=0;j<8;j++)
      sum += siftPoint[16*j+tx] * ptr2[16*j+tx];
  sums[i] = sum;
  __syncthreads();
  if (tx<8)
    sums[i] += sums[i+8];
  __syncthreads();
  if (tx<4)
    sums[i] += sums[i+4];
  __syncthreads();
  if (tx<2)
    sums[i] += sums[i+2];
  __syncthreads();
  if (tx<1)
    sums[i] += sums[i+1];
  __syncthreads();
  if (ty==0) {
    corrData[p1*gridDim.y*16 + blockIdx.y*16 + tx] = sums[16*tx];
    //printf("corr = %.2f\n", sums[16*tx]);
  }
  __syncthreads();
}


__global__ void FindMaxCorr(float *corrData, SiftPoint *sift1, SiftPoint *sift2, int numPts1, int corrWidth, int siftSize)
{
  __shared__ float maxScore[16*16];
  __shared__ float maxScor2[16*16];
  __shared__ int maxIndex[16*16];
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int idx = ty*16 + tx;
  int p1 = blockIdx.x*16 + threadIdx.y;
  p1 = (p1>=numPts1 ? numPts1-1 : p1);
  maxScore[idx] = -1.0f;
  maxScor2[idx] = -1.0f;
  maxIndex[idx] = -1;
  __syncthreads();
  float *corrs = &corrData[p1*corrWidth];
  for (int i=tx;i<corrWidth;i+=16) {
    float val = corrs[i];
    if (val>maxScore[idx]) {
      maxScor2[idx] = maxScore[idx];
      maxScore[idx] = val;
      maxIndex[idx] = i;
    } else if (val>maxScor2[idx])
      maxScor2[idx] = val;
  }
  //if (p1==1)
  //  printf("tx = %d, score = %.2f, scor2 = %.2f, index = %d\n", 
  //	   tx, maxScore[idx], maxScor2[idx], maxIndex[idx]);
  __syncthreads();
  for (int len=8;len>0;len/=2) {
    if (tx<8) {
      float val = maxScore[idx+len];
      int i = maxIndex[idx+len];
      if (val>maxScore[idx]) {
	maxScor2[idx] = maxScore[idx];
	maxScore[idx] = val;
	maxIndex[idx] = i;
      } else if (val>maxScor2[idx])
	maxScor2[idx] = val;
      float va2 = maxScor2[idx+len];
      if (va2>maxScor2[idx])
	maxScor2[idx] = va2;
    }
    __syncthreads();
    //if (p1==1 && tx<len) 
    //  printf("tx = %d, score = %.2f, scor2 = %.2f, index = %d\n", 
    //	     tx, maxScore[idx], maxScor2[idx], maxIndex[idx]);
  }
  if (tx==6)
    sift1[p1].score = maxScore[ty*16];
  if (tx==7)
    sift1[p1].ambiguity = maxScor2[ty*16] / (maxScore[ty*16] + 1e-6);
  if (tx==8)
    sift1[p1].match = maxIndex[ty*16];
  if (tx==9)
    sift1[p1].match_xpos = sift2[maxIndex[ty*16]].xpos;
  if (tx==10)
    sift1[p1].match_ypos = sift2[maxIndex[ty*16]].ypos;
  __syncthreads();
  //if (tx==0)
  //  printf("index = %d/%d, score = %.2f, ambiguity = %.2f, match = %d\n", 
  //	p1, numPts1, sift1[p1].score, sift1[p1].ambiguity, sift1[p1].match);
}




double MatchSiftData(SiftData &data1, SiftData &data2)
{
  if (data1.d_data==NULL || data2.d_data==NULL)
    return 0.0f;
  TimerGPU timer(0);
  int numPts1 = data1.numPts;
  int numPts2 = data2.numPts;
  SiftPoint *sift1 = data1.d_data;
  SiftPoint *sift2 = data2.d_data;
  
  float *d_corrData; 
  int corrWidth = iDivUp(numPts2, 16)*16;
  int corrSize = sizeof(float)*numPts1*corrWidth;
  safeCall(hipMalloc((void **)&d_corrData, corrSize));
  dim3 blocks(numPts1, iDivUp(numPts2, 16));
  dim3 threads(16, 16); // each block: 1 points x 16 points
  MatchSiftPoints<<<blocks, threads>>>(sift1, sift2, d_corrData, numPts1, numPts2);
  safeCall(hipDeviceSynchronize());
  dim3 blocksMax(iDivUp(numPts1, 16));
  dim3 threadsMax(16, 16);
  FindMaxCorr<<<blocksMax, threadsMax>>>(d_corrData, sift1, sift2, numPts1, corrWidth, sizeof(SiftPoint));
  safeCall(hipDeviceSynchronize());
  checkMsg("MatchSiftPoints() execution failed\n");
  safeCall(hipFree(d_corrData));
  if (data1.h_data!=NULL) {
    float *h_ptr = &data1.h_data[0].score;
    float *d_ptr = &data1.d_data[0].score;

    safeCall(hipMemcpy2D(h_ptr, sizeof(SiftPoint), d_ptr, sizeof(SiftPoint), 5*sizeof(float), data1.numPts, hipMemcpyDeviceToHost));
  }

  double gpuTime = timer.read();
  #ifdef VERBOSE
    printf("MatchSiftData time =          %.2f ms\n", gpuTime);
  #endif
  return gpuTime;
}		 

